
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <time.h>
#define BLOCK_SIZE 4
#define div_up(x, y) ( (y) * ( ((x)+(y)-1) / (y) ) )
//Funcion multiplicacion de matric en memoria global (GM)
__global__ void Multiplica_Matrices_GM(float *C, float *A, float *B, int nfil, int ncol)
{
    int idx=blockIdx.x * blockDim.x + threadIdx.x;
    int idy=blockIdx.y * blockDim.y + threadIdx.y;
    int index=idy*ncol+idx;
    if(idx<nfil && idx<ncol)
    {
        float sum=0.0f;
        for(int k=0; k<ncol; k++){
            sum+=A[idy*ncol+k]*B[k*ncol+idx];
        }
        C[index]=sum;
    }
}

int main(int argc, char const *argv[])
{
    clock_t t_ini, t_fin;
    double secs;
    t_ini=clock();
    srand(time(NULL));
    float *A_h, *B_h,*C_h; //punteros a matrices en el Host
    float *A_d, *B_d, *C_d; //punteros a matrices en el Device
    int nfil = 1000;
    int ncol = 1000;
    int N = nfil*ncol; //numero de elementos de la mtriz

    //GPU time
    hipEvent_t start, stop;
    float time;
    size_t size = N * sizeof(float);
    A_h=(float *)malloc(size);//se pide memoria al host
    B_h=(float *)malloc(size);
    C_h=(float *)malloc(size);

    //Inicializamos las matrices a,b en el host
    for(int i=0; i<nfil; i++)
    {
        for(int j=0; j<ncol; j++)
        {
            A_h[i*ncol+j] = (rand() % 4)+1;
            B_h[i*ncol+j] = (rand() % 4)+1;
        }
    }
    
    //Pedimos memoria en el Device
    hipMalloc((void**) &A_d, size);
    hipMalloc((void**) &B_d, size);
    hipMalloc((void**) &C_d, size);

    //Copiamos la matriz a y b del Host al Device
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
    
    //Realizamos el cálculo en el Device
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 n_blocks(div_up(ncol,block_size.x), div_up(nfil,block_size.y));

    Multiplica_Matrices_GM<<<n_blocks, block_size>>> (C_d, A_d, B_d, nfil, ncol);

    //Pasamos el resultado del Device al Host
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    //imprimos el resultado.
    /*printf("\n\n Matriz en cuda C: \n");
    for(int i=0; i<10; i++)
    {
        for(int j=0; j<=10; j++)
        {
            printf("%.2f\t", C_h[i*ncol+j]);
        }
        printf("\n");
    }*/
    //Liberamos la memoria del Host
    free(A_h);
    free(B_h);
    free(C_h);

    //Liberamos la memoria del Device
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    t_fin=clock();
    secs=(double)(t_fin -t_ini)/CLOCKS_PER_SEC;
    printf("%.16g milisegundos\n", secs*1000.0);

    return 0;
}
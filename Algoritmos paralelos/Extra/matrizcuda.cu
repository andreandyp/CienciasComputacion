
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define BLOCK_SIZE 4
#define div_up(x, y) ( (y) * ( ((x)+(y)-1) / (y) ) )
//Funcion multiplicacion de matric en memoria global (GM)
__global__ void Multiplica_Matrices_GM(float *C, float *A, float *B, int nfilA, int ncolA, int nfilB, int ncolB)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    int index = idy*ncolB+idx;

    if(idy < nfilA && idx<ncolB)
    {
        float sum=0.0f;
        for(int k=0; k < ncolA; k++){
            printf("%f x %f\n", A[idy*ncolA+k], B[k*ncolB+idx]);
            sum+= A[idy*ncolA+k]*B[k*ncolB+idx];
        }
        C[index]=sum;
    }
}

int main(int argc, char const *argv[])
{
    clock_t t_ini, t_fin;
    double secs;
    srand(time(NULL));
    float *A_h, *B_h,*C_h; //punteros a matrices en el Host
    float *A_d, *B_d, *C_d; //punteros a matrices en el Device
    int nfilA = 3;
    int ncolA = 2;

    int nfilB = 2;
    int ncolB = 3;

    
    hipEvent_t start, stop;
    A_h=(float *)malloc(nfilA * ncolA);
    B_h=(float *)malloc(nfilB * ncolB);
    C_h=(float *)malloc(nfilA * ncolB);

    for(int i=0; i<nfilA; i++)
    {
        for(int j=0; j<ncolA; j++)
        {
            A_h[i*ncolA+j] = (rand() % 4)+1;
        }
    }


    for(int i=0; i<nfilB; i++)
    {
        for(int j=0; j<ncolB; j++)
        {
            B_h[i*ncolB+j] = (rand() % 4)+1;
        }
    }

    //Aquí se pueden ver las matrices
    /*for(int i=0; i<nfilA; i++)
    {
        for(int j=0; j<ncolA; j++)
        {
            printf("%.2f\t", A_h[i*ncolA+j]);
        }
        printf("\n");
    }

    printf("--------\n");

    for(int i=0; i<nfilB; i++)
    {
        for(int j=0; j<ncolB; j++)
        {
            printf("%.2f\t", B_h[i*ncolB+j]);
        }
        printf("\n");
    }

    printf("--------\n");
    */
    
    size_t sizeA = (nfilA * ncolA) * sizeof(float);
    size_t sizeB = (nfilB * ncolB) * sizeof(float);
    size_t sizeC = (nfilA * ncolB) * sizeof(float);
    
    hipMalloc((void**) &A_d, sizeA);
    hipMalloc((void**) &B_d, sizeB);
    hipMalloc((void**) &C_d, sizeC);

    
    hipMemcpy(A_d, A_h, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, sizeB, hipMemcpyHostToDevice);
    
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 n_blocks(div_up(ncolB,block_size.x), div_up(nfilA,block_size.y));

    t_ini=clock();
    Multiplica_Matrices_GM<<<n_blocks, block_size>>> (C_d, A_d, B_d, nfilA, ncolA, nfilB, ncolB);
    t_fin = clock();

    hipMemcpy(C_h, C_d, sizeC, hipMemcpyDeviceToHost);

    //Aquí se puede ver el resultado de la multiplicación
    /*printf("\n\n Matriz en cuda C: \n");
    for(int i=0; i<nfilA; i++)
    {
        for(int j=0; j<ncolB; j++)
        {
            printf("%.2f\t", C_h[i*ncolB+j]);
        }
        printf("\n");
    }*/
    

    free(A_h);
    free(B_h);
    

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    secs = (double)(t_fin - t_ini)/CLOCKS_PER_SEC;
    printf("%.16g milisegundos\n", secs*1000.0);

    return 0;
}